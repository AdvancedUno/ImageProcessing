#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<>
#include "file.cuh"
#include "hip/hip_runtime.h"

#include <iostream>

#include <hipfft/hipfft.h>

#include "hipblas.h"

#include <stdio.h>

#include <stdlib.h>
#include <nppdefs.h>
#include ""
#include "npp.h"
#include "nppi.h"


__global__ void Inversion(float* ImageNorm, unsigned char* Image, unsigned char* Image2, int Channels, double minValue1, double maxValue1, double minValue2, double maxValue2) {

    int idx = (blockIdx.x +blockIdx.y * gridDim.x) * Channels;

    for (int i = 0; i < Channels; i++) {

		//ImageNorm[idx + i] = ((Image2[idx + i] - minValue1) * (1.0/(maxValue1 - minValue1))) + ((Image[idx + i] - minValue2) * (1.0 / (maxValue2 - minValue2)));
		//ImageNorm[idx + i] = ((Image2[idx + i]) - minValue1 * 255.0 / (maxValue1 - minValue1))*255.0 / (maxValue1 - minValue1) + ((Image[idx + i] ) - minValue2 * 255.0 / (maxValue2 - minValue2)) * 255.0 / (maxValue2 - minValue2);
		ImageNorm[idx*2 + i] = (ImageNorm[idx + i] + Image2[idx + i]);

    }

}

__global__ void NormImage(float* ImageNorm, unsigned char* Image, int Channels) {

	int idx = (blockIdx.x + blockIdx.y * gridDim.x) * Channels;

	for (int i = 0; i < Channels; i++) {

		Image[idx + i] = ImageNorm[idx + i]/50;
		//Image[idx + i] = 255;

	}

}


void TEMP:: Image_Inversion(float* Input_Norm, unsigned char* Input_Image, unsigned char* Input_image2, int Height, int Width, int Channels, bool bLast, double minValue1, double maxValue1, double minValue2, double maxValue2)
{
	unsigned char* Dev_Input_Image = NULL;
	unsigned char* Dev_Input_Image1 = NULL;
	float* Dev_Input_Norm = NULL;


	//allocate the memory in gpu
	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);


	hipMalloc((void**)&Dev_Input_Image1, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Image1, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);



	hipMalloc((void**)&Dev_Input_Norm, Height * Width * Channels);

	//copy data from CPU to GPU
	hipMemcpy(Dev_Input_Norm, Input_Norm, Height * Width * Channels, hipMemcpyHostToDevice);



	dim3 Grid_Image(Width, Height);

	Inversion << <Grid_Image, 1 >> > (Dev_Input_Norm, Dev_Input_Image, Dev_Input_Image1, Channels, minValue1, maxValue1, minValue2, maxValue2);

	if (bLast) {
		NormImage << <Grid_Image, 1 >> > (Dev_Input_Norm, Dev_Input_Image, Channels);
	}

	//copy processed data back to cpu from gpu
	hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);
	hipMemcpy(Input_Norm, Dev_Input_Norm, Height * Width * Channels, hipMemcpyDeviceToHost);


	

	//free gpu mempry
	hipFree(Dev_Input_Image);
	hipFree(Dev_Input_Image1);
	hipFree(Dev_Input_Norm);
}

//__global__ void sum(int a, int b,int* c) {
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	c[tid] = a + b;
//}
//
//int TEMP::sum_cuda(int a, int b, int* c) {
//
//	int* f;
//	
//	hipMalloc((void**)&f, sizeof(int) * 1);
//	hipMemcpy(f, c, sizeof(int) * 1, hipMemcpyHostToDevice);
//
//	sum << <1,1 >> > (a, b, f);
//
//	hipMemcpy(c, f, sizeof(int) * 1, hipMemcpyDeviceToHost);
//
//	hipFree(f);
//
//	return true;
//
//
//}


